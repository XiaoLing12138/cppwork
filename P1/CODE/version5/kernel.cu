#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "struct.h"

#include <iostream>
#include <cstdio>

const int threads_sum = 16;
const int BLOCK_SIZE = threads_sum;
const int ROW = 2000;
const int COL = 50000;

__global__ void Kernel(float* devic_a, float* devic_b, float* devic_c,
						size_t n, size_t m)
{
	__shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
	__shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

	size_t bx = blockIdx.x;
	size_t by = blockIdx.y;

	size_t tx = threadIdx.x;
	size_t ty = threadIdx.y;

	size_t Row = by * BLOCK_SIZE + ty;
	size_t Col = bx * BLOCK_SIZE + tx;

	float cSum = 0.0;

	for (size_t i = 0; i < n; i += BLOCK_SIZE)
	{

		As[ty][tx] = devic_a[Row * n + i + tx];
		Bs[ty][tx] = devic_b[(i + ty) * m + Col];

		__syncthreads();
		//Synchronization is required before the calculation
		//for each thread in the block has finished reading

		for (size_t k = 0; k < BLOCK_SIZE; k++)
		{
			cSum += As[ty][k] * Bs[k][tx];
		}

		__syncthreads();
		//Synchronize before going to the next calculation
		//to avoid changing the values in the Shared memory
	}
	devic_c[m * Row + Col] = cSum;
}

void matrixMul(Matrix a, Matrix b, Matrix c, dim3 grid, dim3 block);

int main()
{
	Matrix a, b, c;

	ios::sync_with_stdio(false);

	a.rows = ROW;
	a.cols = COL;
	b.rows = COL;
	b.cols = ROW;
	a.step = a.cols;
	b.step = b.cols;
	a.data = (float*)malloc(sizeof(float) * a.rows * a.cols);
	b.data = (float*)malloc(sizeof(float) * b.rows * b.cols);
	if (a.data != 0)
	{
		memset(a.data, 0, sizeof(float) * a.rows * a.cols);
	}
	if (b.data != 0)
	{
		memset(b.data, 0, sizeof(float) * b.rows * b.cols);
	}
	a.data[0] = (float)1.0;
	b.data[0] = (float)1.0;
	a.data[a.rows * a.cols - 1] = (float)2.0;
	b.data[b.rows * b.cols - 1] = (float)2.0;

    dim3 grid(a.rows / threads_sum, b.cols / threads_sum);
    dim3 block(threads_sum, threads_sum);

	if (a.cols != b.rows)
	{
		printf("Your matrixes do not match!");
		return 0;
	}

	c.rows = ROW;
	c.cols = ROW;
	c.step = c.cols;
	c.data = (float*)malloc(sizeof(float) * a.rows * b.cols);

	matrixMul(a, b, c, grid, block);

	free(a.data);
	free(b.data);
	free(c.data);
	return 0;
}

void matrixMul(Matrix a, Matrix b, Matrix c, dim3 grid, dim3 block)
{
	float* devic_a;
	float* devic_b;
	float* devic_c;

	hipEvent_t start, stop;
	float running_time;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipMalloc((void**)&devic_a, a.rows * a.cols * sizeof(float));
	hipMalloc((void**)&devic_b, b.rows * b.cols * sizeof(float));
	hipMalloc((void**)&devic_c, c.rows * c.cols * sizeof(float));

	hipEventRecord(start, 0);

	hipMemcpy(devic_a, a.data, a.rows * a.cols * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(devic_b, b.data, b.rows * b.cols * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(devic_c, c.data, c.rows * c.cols * sizeof(float), hipMemcpyHostToDevice);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&running_time, start, stop);

	cout << "Writing memcpy time:" << running_time << "ms" << endl;

	hipEventRecord(start, 0);

	Kernel << <grid, block >> > (devic_a, devic_b, devic_c, a.cols, b.cols);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&running_time, start, stop);
	cout << "Calculating time2:" << running_time << "ms" << endl;

	hipEventRecord(start, 0);

	hipMemcpy(c.data, devic_c, c.rows * c.cols * sizeof(float), hipMemcpyDeviceToHost);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&running_time, start, stop);
	cout << "Writing back time:" << running_time << "ms" << endl;

	printf("%0.2f %0.2f\n", c.data[0], c.data[c.rows * c.cols - 1]);

	hipFree(devic_a);
	hipFree(devic_b);
	hipFree(devic_c);
}
