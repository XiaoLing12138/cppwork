#include "hip/hip_runtime.h"
#include ""
#include "struct.h"
#include "hipblas.h"

#include <iostream>
#include <cstdio>

const int AROW = 2000;
const int ACOL = 50000;
const int BROW = 50000;
const int BCOL = 2000;

void matrixMul(Matrix a, Matrix b, Matrix c);

int main()
{
	Matrix a, b, c;

	ios::sync_with_stdio(false);

	a.rows = AROW;
	a.cols = ACOL;
	b.rows = BROW;
	b.cols = BCOL;
	a.step = a.cols;
	b.step = b.cols;
	a.data = (float*)malloc(sizeof(float) * a.rows * a.cols);
	b.data = (float*)malloc(sizeof(float) * b.rows * b.cols);
	if (a.data != 0)
	{
		memset(a.data, 0, sizeof(float) * a.rows * a.cols);
	}
	if (b.data != 0)
	{
		memset(b.data, 0, sizeof(float) * b.rows * b.cols);
	}
	a.data[0] = 1.0f;
	b.data[0] = 1.0f;
	a.data[a.rows * a.cols - 1] = 2.0f;
	b.data[b.rows * b.cols - 1] = 2.0f;

	if (a.cols != b.rows)
	{
		printf("Your matrixes do not match!");
		return 0;
	}

	c.rows = AROW;
	c.cols = BCOL;
	c.step = c.cols;
	c.data = (float*)malloc(sizeof(float) * c.rows * c.cols);
	if (c.data != 0)
	{
		memset(c.data, 0, sizeof(float) * c.rows * c.cols);
	}

	matrixMul(a, b, c);

	free(a.data);
	free(b.data);
	free(c.data);
	return 0;
}

void matrixMul(Matrix a, Matrix b, Matrix c)
{
	float* cuda_a;
	float* cuda_b;
	float* cuda_c;

	hipEvent_t start, stop;
	float running_time;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipMalloc((void**)&cuda_a, a.rows * a.cols * sizeof(float));
	hipMalloc((void**)&cuda_b, b.rows * b.cols * sizeof(float));
	hipMalloc((void**)&cuda_c, c.rows * c.cols * sizeof(float));

	hipblasHandle_t handle;

	hipblasCreate(&handle);
	
	hipblasSetMatrix(AROW, ACOL, sizeof(*a.data), a.data, AROW, cuda_a, AROW);
	hipblasSetMatrix(BROW, BCOL, sizeof(*b.data), b.data, BROW, cuda_b, BROW);
	hipblasSetMatrix(AROW, BCOL, sizeof(*c.data), c.data, AROW, cuda_c, BCOL);
	
	float alpha = 1, beta = 0;

	hipEventRecord(start, 0);

	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
		AROW, BCOL, ACOL, &alpha, cuda_a,
		AROW, cuda_b, BROW, &beta, cuda_c, BCOL);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&running_time, start, stop);

	cout << "cuBLAS time: " << running_time << "ms" << endl;

	hipblasGetMatrix(AROW, BCOL, sizeof(*c.data), cuda_c, AROW, c.data, BCOL);

	printf("%0.2f %0.2f\n", c.data[0], c.data[c.rows * c.cols - 1]);

	hipFree(cuda_a);
	hipFree(cuda_b);
	hipFree(cuda_c);
}
