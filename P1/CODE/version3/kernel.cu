#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "struct.h"

#include <iostream>
#include <cstdio>
#include <ctime>
#include <chrono>

const int threads_sum = 16;

__global__ void Kernel(float* devic_a, float* devic_b, float* devic_c,
	size_t n, size_t m)
{
	size_t x = blockIdx.x * blockDim.x + threadIdx.x;
	size_t y = blockIdx.y * blockDim.y + threadIdx.y;
	size_t offset = x * blockDim.x * gridDim.x + y;

	float ans = 0;
	for (size_t k = 0; k < n; k++)
	{
		ans += devic_a[x * n + k] * devic_b[k * m + y];
	}
	devic_c[offset] = ans;
}

void matrixMul(Matrix a, Matrix b, Matrix c, dim3 grid, dim3 block);

int main()
{
	Matrix a, b, c;

	ios::sync_with_stdio(false);

	a.rows = 2000;
	a.cols = 50000;
	b.rows = 50000;
	b.cols = 2000;
	a.step = a.cols;
	b.step = b.cols;
	a.data = (float*)malloc(sizeof(float) * a.rows * a.cols);
	b.data = (float*)malloc(sizeof(float) * b.rows * b.cols);
	if (a.data != 0)
	{
		memset(a.data, 0, sizeof(float) * a.rows * a.cols);
	}
	if (b.data != 0)
	{
		memset(b.data, 0, sizeof(float) * b.rows * b.cols);
	}
	a.data[0] = (float)1.0;
	b.data[0] = (float)1.0;
	a.data[a.rows * a.cols - 1] = (float)2.0;
	b.data[b.rows * b.cols - 1] = (float)2.0;

	dim3 grid(a.rows / threads_sum, b.cols / threads_sum);
	dim3 block(threads_sum, threads_sum);

	if (a.cols != b.rows)
	{
		printf("Your matrixes do not match!");
		return 0;
	}

	c.rows = 2000;
	c.cols = 2000;
	c.step = 2000;
	c.data = (float*)malloc(sizeof(float) * a.rows * b.cols);

	matrixMul(a, b, c, grid, block);

	free(a.data);
	free(b.data);
	free(c.data);

	return 0;
}

void matrixMul(Matrix a, Matrix b, Matrix c, dim3 grid, dim3 block)
{
	float* devic_a;
	float* devic_b;
	float* devic_c;

	hipEvent_t start, stop;
	float running_time;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start, 0);

	hipMalloc((void**)&devic_a, a.rows * a.cols * sizeof(float));
	hipMalloc((void**)&devic_b, b.rows * b.cols * sizeof(float));
	hipMalloc((void**)&devic_c, c.rows * c.cols * sizeof(float));

	hipMemcpy(devic_a, a.data, a.rows * a.cols * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(devic_b, b.data, b.rows * b.cols * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(devic_c, c.data, c.rows * c.cols * sizeof(float), hipMemcpyHostToDevice);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&running_time, start, stop);

	cout << "Writing memcpy time:" << running_time << "ms" << endl;

	hipEventRecord(start, 0);

	Kernel << <grid, block >> > (devic_a, devic_b, devic_c, a.cols, b.cols);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&running_time, start, stop);
	cout << "Calculating time2:" << running_time << "ms" << endl;

	hipEventRecord(start, 0);

	hipMemcpy(c.data, devic_c, c.rows * c.cols * sizeof(float), hipMemcpyDeviceToHost);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&running_time, start, stop);
	cout << "Writing back time:" << running_time << "ms" << endl;

	printf("%0.2f %0.2f\n", c.data[0], c.data[c.rows * c.cols - 1]);

	hipFree(devic_a);
	hipFree(devic_b);
	hipFree(devic_c);
}