#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "struct.h"

#include <iostream>
#include <cstdio>
#include <ctime>
#include <chrono>

const int threads_sum = 16;

__global__ void Kernel(float* devic_a, float* devic_b, float* devic_c,
						size_t n, size_t m)
{
	size_t x = size_t(blockIdx.x) * blockDim.x + threadIdx.x;
	size_t y = size_t(blockIdx.y) * blockDim.y + threadIdx.y;
	size_t offset = x * blockDim.x * gridDim.x + y;

	float ans = 0;
	for (size_t i = 0; i < n; i++)
	{
		ans += devic_a[x * n + i] * devic_b[i * m + y];
	}
	devic_c[offset] = ans;
}

void matrixMul(Matrix a, Matrix b, Matrix c, dim3 grid, dim3 block);

int main()
{
	Matrix a, b, c;

	a.rows = 2000;
	a.cols = 50000;
	b.rows = 50000;
	b.cols = 2000;
	a.step = a.cols;
	b.step = b.cols;
	a.data = (float*)malloc(sizeof(float) * a.rows * a.cols);
	b.data = (float*)malloc(sizeof(float) * b.rows * b.cols);
	if (a.data != 0)
	{
		memset(a.data, 0, sizeof(float) * a.rows * a.cols);
	}
	if (b.data != 0)
	{
		memset(b.data, 0, sizeof(float) * b.rows * b.cols);
	}
	a.data[0] = (float)1.0;
	b.data[0] = (float)1.0;
	a.data[a.rows * a.cols - 1] = (float)2.0;
	b.data[b.rows * b.cols - 1] = (float)2.0;

    dim3 grid(a.rows / threads_sum, b.cols / threads_sum);
    dim3 block(threads_sum, threads_sum);

	if (a.cols != b.rows)
	{
		printf("Your matrixes do not match!");
		return 0;
	}

	c.rows = 2000;
	c.cols = 2000;
	c.step = 2000;
	c.data = (float*)malloc(sizeof(float) * a.rows * b.cols);

	matrixMul(a, b, c, grid, block);

	free(a.data);
	free(b.data);
	free(c.data);

	return 0;
}

void matrixMul(Matrix a, Matrix b, Matrix c, dim3 grid, dim3 block)
{
	float* devic_a;
	float* devic_b;
	float* devic_c;

	hipMalloc((void**)&devic_a, a.rows * a.cols * sizeof(float));
	hipMalloc((void**)&devic_b, b.rows * b.cols * sizeof(float));
	hipMalloc((void**)&devic_c, c.rows * c.cols * sizeof(float));

	hipMemcpy(devic_a, a.data, a.rows * a.cols * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(devic_b, b.data, b.rows * b.cols * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(devic_c, c.data, c.rows * c.cols * sizeof(float), hipMemcpyHostToDevice);

	chrono::steady_clock::time_point start = chrono::steady_clock::now();

	Kernel << <grid, block >> > (devic_a, devic_b, devic_c, a.cols, b.cols);

	chrono::steady_clock::time_point end = chrono::steady_clock::now();

	hipMemcpy(c.data, devic_c, c.rows * c.cols * sizeof(float), hipMemcpyDeviceToHost);

	printf("%0.2f %0.2f\n", c.data[0], c.data[c.rows * c.cols - 1]);

	cout << "CUDA takes "
		<< chrono::duration_cast<chrono::microseconds>(end - start).count()
		<< "us.\n";

	hipFree(devic_a);
	hipFree(devic_b);
	hipFree(devic_c);
}