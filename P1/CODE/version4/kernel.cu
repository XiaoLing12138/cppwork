#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "struct.h"

#include <iostream>
#include <cstdio>
#include <ctime>
#include <chrono>

const int threads_sum = 16;
const int BLOCK_SIZE = threads_sum;

__global__ void Kernel(float* devic_a, float* devic_b, float* devic_c,
						size_t n, size_t m)
{
	__shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
	__shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

	size_t bx = blockIdx.x;
	size_t by = blockIdx.y;

	size_t tx = threadIdx.x;
	size_t ty = threadIdx.y;

	size_t Row = by * BLOCK_SIZE + ty;
	size_t Col = bx * BLOCK_SIZE + tx;

	float cSum = 0.0f;

	for (size_t i = 0; i < n; i += BLOCK_SIZE)
	{

		As[ty][tx] = devic_a[Row * n + i + tx];
		Bs[ty][tx] = devic_b[(i + ty) * m + Col];

		__syncthreads();
		//Synchronization is required before the calculation, 
		//for each thread in the block has finished reading

		for (size_t k = 0; k < BLOCK_SIZE; k++)
		{
			cSum += As[ty][k] * Bs[k][tx];
		}

		__syncthreads();
		//Synchronize before going to the next calculation 
		//to avoid changing the values in the Shared memory
	}

	devic_c[m * Row + Col] = cSum;
}

void matrixMul(Matrix a, Matrix b, Matrix c, dim3 grid, dim3 block);

int main()
{
	Matrix a, b, c;

	a.rows = 2000;
	a.cols = 50000;
	b.rows = 50000;
	b.cols = 2000;
	a.step = a.cols;
	b.step = b.cols;
	a.data = (float*)malloc(sizeof(float) * a.rows * a.cols);
	b.data = (float*)malloc(sizeof(float) * b.rows * b.cols);
	if (a.data != 0)
	{
		memset(a.data, 0, sizeof(float) * a.rows * a.cols);
	}
	if (b.data != 0)
	{
		memset(b.data, 0, sizeof(float) * b.rows * b.cols);
	}
	a.data[0] = (float)1.0;
	b.data[0] = (float)1.0;
	a.data[a.rows * a.cols - 1] = (float)2.0;
	b.data[b.rows * b.cols - 1] = (float)2.0;

    dim3 grid(a.rows / threads_sum, b.cols / threads_sum);
    dim3 block(threads_sum, threads_sum);

	if (a.cols != b.rows)
	{
		printf("Your matrixes do not match!");
		return 0;
	}

	c.rows = 2000;
	c.cols = 2000;
	c.step = 2000;
	c.data = (float*)malloc(sizeof(float) * a.rows * b.cols);

	matrixMul(a, b, c, grid, block);

	free(a.data);
	free(b.data);
	free(c.data);

	return 0;
}

void matrixMul(Matrix a, Matrix b, Matrix c, dim3 grid, dim3 block)
{
	float* devic_a;
	float* devic_b;
	float* devic_c;

	chrono::steady_clock::time_point startR = chrono::steady_clock::now();

	hipMalloc((void**)&devic_a, a.rows * a.cols * sizeof(float));
	hipMalloc((void**)&devic_b, b.rows * b.cols * sizeof(float));
	hipMalloc((void**)&devic_c, c.rows * c.cols * sizeof(float));

	hipMemcpy(devic_a, a.data, a.rows * a.cols * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(devic_b, b.data, b.rows * b.cols * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(devic_c, c.data, c.rows * c.cols * sizeof(float), hipMemcpyHostToDevice);

	chrono::steady_clock::time_point start = chrono::steady_clock::now();

	Kernel << <grid, block >> > (devic_a, devic_b, devic_c, a.cols, b.cols);

	chrono::steady_clock::time_point end = chrono::steady_clock::now();

	hipMemcpy(c.data, devic_c, c.rows * c.cols * sizeof(float), hipMemcpyDeviceToHost);

	chrono::steady_clock::time_point endR = chrono::steady_clock::now();

	printf("%0.2f %0.2f\n", c.data[0], c.data[c.rows * c.cols - 1]);

	cout << "CUDA takes "
		<< chrono::duration_cast<chrono::microseconds>(end - start).count()
		<< "us to count.\n";

	cout << "CUDA takes "
		<< chrono::duration_cast<chrono::seconds>(endR - startR).count()
		<< "s to write memory.\n";

	hipFree(devic_a);
	hipFree(devic_b);
	hipFree(devic_c);
}